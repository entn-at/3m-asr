#include "hip/hip_runtime.h"
#include "moe_cuda_kernel.h"

#include <cstdio>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <c10/cuda/CUDAGuard.h>

#include "timer.hh"

#include "cublas_wrapper.h"
#include "cuda_stream_manager.h"

#define CEIL(_x_,_y_) (((_x_)-1)/(_y_)+1)

template <typename scalar_t>
__global__
void generate_ptr_offset_kernel(size_t n, const scalar_t* base, size_t stride,
		const long* offset, const scalar_t** ptrs) { 
	size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < n) {
		ptrs[idx] = base + stride * offset[idx];
	}
}

template <typename scalar_t>
__global__
void batch_scatter_kernel(size_t wid, const long* pos, 
		const scalar_t* inbuf, scalar_t* oubuf) { 
	inbuf += wid * pos[blockIdx.x];
	oubuf += wid * blockIdx.x;
	for (int i = threadIdx.x; i < wid; i += blockDim.x) {
		oubuf[i] = inbuf[i];
	}
}

void moe_cuda_expert_count_impl(
        const int* d_gate,
		int* expert_count,
		int* d_pos,
		const size_t num_expert,
        const size_t batch_size) {
    int *gate = new int[batch_size];
	int *expert_ptr = new int[num_expert];
	memset(expert_count, 0, sizeof(int) * num_expert);

	checkCudaErrors(hipMemcpy(gate, d_gate, sizeof(int) * batch_size,
				hipMemcpyDeviceToHost));

	for (int i = 0; i < batch_size; ++i) {
		++expert_count[gate[i]];
	}
	expert_ptr[0] = 0;
	for (int i = 1; i < num_expert; ++i) {
		expert_ptr[i] = expert_ptr[i - 1] + expert_count[i - 1];
	}

	int *pos = new int[batch_size];

	for (int i = 0; i < batch_size; ++i) {
		pos[i] = expert_ptr[gate[i]]++;
	}
	for (int i = num_expert - 1; i > 0; --i) {
		expert_ptr[i] = expert_ptr[i - 1];
	}
	expert_ptr[0] = 0;
	checkCudaErrors(hipMemcpy(d_pos, pos, sizeof(int) * batch_size,
				hipMemcpyHostToDevice));
	delete [] gate;
	delete [] expert_ptr;
}

template <typename scalar_t>
void moe_cuda_local_scatter_impl(
        const scalar_t* input,
		const long* d_pos,
		scalar_t* input_buf,
		const long batch_size,
		const long in_feat, 
		CudaStreamManager* smgr) {
	batch_scatter_kernel<scalar_t>
		<<<batch_size, 256, 0, smgr->stream(0)>>>(in_feat, d_pos, input,
				input_buf); 
	smgr->sync(1);
}

template <typename scalar_t>
__global__
void batch_gather_kernel(size_t wid, const long* pos, 
		const scalar_t* inbuf, scalar_t* oubuf) { 
	inbuf += wid * blockIdx.x;
	oubuf += wid * pos[blockIdx.x];
	for (int i = threadIdx.x; i < wid; i += blockDim.x) {
		oubuf[i] = inbuf[i];
	}
}

template <typename scalar_t>
void moe_cuda_local_gather_impl(
        const scalar_t* output_buf,
		const long* d_pos,
		scalar_t* output,
		const size_t batch_size,
		const size_t out_feat,
		CudaStreamManager* smgr) {
	batch_gather_kernel<scalar_t>
		<<<batch_size, 256, 0, smgr->stream(0)>>>(out_feat, d_pos, output_buf,
				output); 
	smgr->sync(1);
}

template <typename scalar_t>
void moe_cuda_forward_impl(
        const scalar_t* input_buf,
        const scalar_t* weight,
		const long* expert_count,
        scalar_t* output_buf,
        const size_t in_feat,
        const size_t out_feat,
        const size_t num_expert,
		CudaStreamManager* smgr,
        int capacity, bool training) {
	scalar_t alpha = 1, beta = 0; 
    long n_samples = 0;
	for (int i = 0, ptr = 0; i < num_expert; ++i) {
		if (expert_count[i] == 0) {
			continue;
		}
        n_samples = expert_count[i];
        if (capacity > 0 && training && n_samples > capacity) {
            n_samples = capacity;
        }
		// Use T(B) x T(A) = T(C) to produce row-major C
		checkCudaErrors(cublasXgemm(
				smgr->handle(i),
				HIPBLAS_OP_T,
				HIPBLAS_OP_N,
				out_feat, n_samples, in_feat,
				&alpha,
				weight + i * in_feat * out_feat, in_feat,
				input_buf + ptr * in_feat, in_feat,
				&beta,
				output_buf + out_feat * ptr, out_feat
				));

		ptr += expert_count[i];
	}
	smgr->sync(num_expert);
}

template <typename scalar_t>
void moe_cuda_backward_impl(
        const scalar_t* grad_output_buf,
        const scalar_t* input_buf,
		const scalar_t* weight,
		const long* expert_count,
        scalar_t* grad_input_buf,
        scalar_t* grad_weight,
        const size_t batch_size,
        const size_t in_feat,
        const size_t out_feat,
        const size_t num_expert,
		CudaStreamManager* smgr,
        int capacity, bool training) {
    scalar_t alpha = 1, beta = 0;
    long n_samples = 0;
	for (int i = 0, ptr = 0; i < num_expert; ++i) {
		if (expert_count[i] == 0) {
			hipMemset(grad_weight + i * in_feat * out_feat, 0, 
					sizeof(scalar_t) * in_feat * out_feat);
			continue;
		}
		// Use T(B) x T(A) = T(C) to produce row-major C

        n_samples = expert_count[i];
        if (capacity > 0 && training && n_samples > capacity) {
            n_samples = capacity;
        }
		// Backward input: g_i = w @ g_o
		checkCudaErrors(cublasXgemm(
				smgr->handle(i),
				HIPBLAS_OP_N,
				HIPBLAS_OP_N,
				in_feat, n_samples, out_feat,
				&alpha,
				weight + i * in_feat * out_feat, in_feat,
				grad_output_buf + ptr * out_feat, out_feat,
				&beta,
				grad_input_buf + in_feat * ptr, in_feat
				));

		// Backward weight: g_w = i @ g_o
		checkCudaErrors(cublasXgemm(
				smgr->handle(i),
				HIPBLAS_OP_N,
				HIPBLAS_OP_T,
				in_feat, out_feat, n_samples,
				&alpha,
				input_buf + in_feat * ptr, in_feat,
				grad_output_buf + ptr * out_feat, out_feat,
				&beta,
				grad_weight + i * in_feat * out_feat, in_feat
				));

		ptr += expert_count[i];
	}
	smgr->sync(num_expert);
}


std::vector<torch::Tensor> moe_cuda_expert_count(
		torch::Tensor gate, 
		size_t num_expert) {
	const auto batch_size = gate.size(0);

	auto ec_options = torch::TensorOptions().dtype(torch::kInt32);
	auto expert_count = torch::empty(num_expert, ec_options);

	auto pos_options = torch::TensorOptions()
		.device(gate.device())
		.dtype(torch::kInt32);
	auto pos = torch::empty(batch_size, pos_options);
	moe_cuda_expert_count_impl(
			gate.data_ptr<int>(),
			expert_count.data_ptr<int>(),
			pos.data_ptr<int>(),
			num_expert,
			batch_size);

	return {expert_count, pos};
}

std::vector<torch::Tensor> moe_cuda_local_scatter(
    torch::Tensor input,
	torch::Tensor pos) {
	auto smgr = getCudaStreamManager(input.device().index());
	const auto batch_size = pos.size(0);
    const auto in_feat = input.size(1);

	auto opt = torch::TensorOptions()
		.dtype(input.dtype())
		.device(input.device());
	auto input_buf = torch::empty({batch_size, in_feat}, opt);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "moe_local_scatter_cuda", 
			([&] {
		moe_cuda_local_scatter_impl<scalar_t>(
			input.data_ptr<scalar_t>(),
			pos.data_ptr<long>(),
			input_buf.data_ptr<scalar_t>(),
			batch_size,
			in_feat,
			smgr);
	}));
	return {input_buf,};
}

std::vector<torch::Tensor> moe_cuda_local_gather(
	torch::Tensor output_buf,
	torch::Tensor pos) {
	auto smgr = getCudaStreamManager(output_buf.device().index());
	const auto batch_size = pos.size(0);
    const auto out_feat = output_buf.size(1);

	auto opt = torch::TensorOptions()
		.dtype(output_buf.dtype())
		.device(output_buf.device());
	auto output = torch::empty({batch_size, out_feat}, opt);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(output_buf.scalar_type(), "moe_local_gather_cuda", 
			([&] {
		moe_cuda_local_gather_impl<scalar_t>(
			output_buf.data_ptr<scalar_t>(),
			pos.data_ptr<long>(),
			output.data_ptr<scalar_t>(),
			batch_size,
			out_feat,
			smgr);
	}));
	return {output,};
}

std::vector<torch::Tensor> moe_cuda_forward(
        torch::Tensor input_buf,
        torch::Tensor weight,
		torch::Tensor expert_count,
        int capacity, bool training
		) {
	auto smgr = getCudaStreamManager(input_buf.device().index());
	const auto batch_size = input_buf.size(0);
    const auto num_expert = weight.size(0);
    const auto out_feat = weight.size(1);
    const auto in_feat = weight.size(2);
            
#ifdef MOE_DEBUG
    printf("[forward] expert=%ld, in_feat (d_model)=%ld, out_feat (d_ffn)=%ld\n", 
			num_expert, in_feat, out_feat);
#endif
	auto out_options = torch::TensorOptions()
		.device(input_buf.device())
		.dtype(input_buf.dtype());
    // auto output = torch::empty({batch_size, out_feat}, out_options);
    auto output = torch::zeros({batch_size, out_feat}, out_options);
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input_buf.scalar_type(), "moe_forward_cuda", 
			([&] {
		moe_cuda_forward_impl<scalar_t>(
			input_buf.data_ptr<scalar_t>(),
			weight.data_ptr<scalar_t>(),
			expert_count.data_ptr<long>(),
			output.data_ptr<scalar_t>(),
			in_feat,
			out_feat,
			num_expert,
			smgr,
            capacity,
            training
		);
    }));
    
    return {output, };           
}

std::vector<torch::Tensor> moe_cuda_backward(
    torch::Tensor grad_output_buf, // [batch_size x out_feat]
    torch::Tensor input_buf, // [batch_size x out_feat]
    torch::Tensor weight, // [num_expert x out_feat x in_feat]
	torch::Tensor expert_count,
    int capacity, bool training
) {
	auto smgr = getCudaStreamManager(input_buf.device().index());
    const auto batch_size = input_buf.size(0);
    const auto num_expert = weight.size(0);
    const auto out_feat = weight.size(1);
    const auto in_feat = weight.size(2);

#ifdef MOE_DEBUG
    printf("[backward] b=%ld, expert=%ld, in_feat (d_model)=%ld, "
			"out_feat (d_ffn)=%ld\n",
			batch_size, num_expert, in_feat, out_feat);
#endif

    // auto grad_input_buf = grad_output_buf.new_empty({batch_size, in_feat}); 
    auto grad_input_buf = grad_output_buf.new_zeros({batch_size, in_feat}); 
    // auto grad_weight = grad_output_buf.new_empty({num_expert, out_feat, in_feat});
    auto grad_weight = grad_output_buf.new_zeros({num_expert, out_feat, in_feat});

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input_buf.scalar_type(), "moe_cuda_backward", ([&] {
        moe_cuda_backward_impl<scalar_t>(
            grad_output_buf.data_ptr<scalar_t>(),
            input_buf.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
			expert_count.data_ptr<long>(),
            grad_input_buf.data_ptr<scalar_t>(),
            grad_weight.data_ptr<scalar_t>(),
            batch_size,
            in_feat,
            out_feat,
            num_expert,
			smgr,
            capacity,
            training
        );
    }));

    return {grad_input_buf, grad_weight};
}
